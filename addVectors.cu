#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include ""
#include "addVectors.cuh"
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        c[i] = a[i] + b[i];
    }
}

#define SIZE 1024

int addConstWrapper() {
    return addWrapper(SIZE);
}

int addWrapper(int arraySize) {
    size_t size = arraySize * sizeof(int);
    // Allocate the host input vector A
    int* h_A = (int*)malloc(size);

    // Allocate the host input vector B
    int* h_B = (int*)malloc(size);

    // Allocate the host output vector C
    int* h_C = (int*)malloc(size);

    // Allocate the assertion vector C
    int* h_C_ass = (int*)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < arraySize; ++i)
    {
        h_A[i] = rand();
        h_B[i] = rand();
        h_C_ass[i] = h_A[i] + h_B[i];
    }

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(h_C, h_A, h_B, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    // Assert
    bool check = true;
    for (int i = 0; i < arraySize; ++i)
    {
        if (h_C_ass[i] != h_C[i]) {
            check = false;
            printf("Failed on index %d. Asserted: %d. From kernel: %d", i, h_C_ass[i], h_C[i]);
        };
    }
    if (check == true) {
        printf("Assertion succeeded!");
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // ����������� � ������� �����������
    unsigned int start_time = clock();

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    int threadsPerBlock = std::min((unsigned int)1024, size);
    int blocksPerGrid = std::max((unsigned int)1, (size + threadsPerBlock - 1) / threadsPerBlock);
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    addKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    unsigned int end_time = clock();
    unsigned int search_time = end_time - start_time;
    printf("����� ����������: %d ��.", search_time);

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}