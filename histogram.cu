#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <cstdlib> // For rand() function
#include <ctime>   // For time() function
#include "hip/hip_runtime.h"
#include ""
#include <cassert>

#define BIN_COUNT 256
#define SHARED_BIN_COUNT 32
#define THREADS 256
#define BLOCKS 1000


unsigned char* generateRandomString(int size) {
    std::srand(std::time(0));
    size_t dataSize = size * sizeof(unsigned char);

    unsigned char* randomString = (unsigned char*)malloc(dataSize);

    for (int i = 0; i < size; ++i) {
        unsigned char randomChar = (unsigned char)(std::rand() % BIN_COUNT);
        //unsigned char randomChar = (unsigned char)(32);
        randomString[i] = randomChar;
    }

    return randomString;
}

__global__ void histogramKernel(unsigned char* d_hist_data,
    long long size, unsigned long long* d_bin_data)
{
    __shared__ unsigned int d_bin_data_shared[BIN_COUNT];
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int step = blockDim.x * gridDim.x;

    d_bin_data_shared[threadIdx.x] = 0;

    __syncthreads();

    
    while (i < size) {
        atomicAdd(&d_bin_data_shared[d_hist_data[i]], 1);
        i += step;
    }

    __syncthreads();

    atomicAdd(&d_bin_data[threadIdx.x], d_bin_data_shared[threadIdx.x]);
}

void myhistogramCPU(unsigned char* h_hist_data, unsigned long long* h_bin_data, unsigned int N) {
    clock_t start_time = clock();

    for (unsigned int i = 0; i < BIN_COUNT; i++) h_bin_data[i] = 0;
    for (unsigned int i = 0; i < N; i++) {
        unsigned char data = h_hist_data[i];
        h_bin_data[data]++;
    }

    clock_t end_time = clock();
    double search_time = (double)(end_time - start_time);
    printf("Время выполнения CPU: %f мс.\n", search_time);
}

hipError_t histogramWithCuda(unsigned char* h_hist_data, unsigned long long* h_bin_data_GPU, unsigned long long size) {
    unsigned char* d_hist_data;
    unsigned long long* d_bin_data;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&d_hist_data, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_bin_data, BIN_COUNT * sizeof(long long));
    hipMemset(d_bin_data, 0, BIN_COUNT * sizeof(long long));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Высчитываем с момента копирования
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    cudaStatus = hipMemcpy(d_hist_data, h_hist_data, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! %d");
        printf("%d", cudaStatus);
        goto Error;
    }

    histogramKernel << <size / 256,
        256 >> > (
            d_hist_data, size, d_bin_data);

    cudaStatus = hipMemcpy(h_bin_data_GPU, d_bin_data, BIN_COUNT * sizeof(long long), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        printf("%d", cudaStatus);
        goto Error;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "histogramKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching histogramKernel!\n", cudaStatus);
        goto Error;
    }

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Время выполнения GPU: %f мс.", milliseconds);


Error:
    hipFree(d_hist_data);
    hipFree(d_bin_data);

    return cudaStatus;
}

int histogramWrapper(unsigned int size) {
    unsigned char* h_hist_data;
    unsigned long long* h_bin_data_CPU, * h_bin_data_GPU;

    h_hist_data = generateRandomString(size);
    /*for (int i = 0; i < size; i++)
    {
        if ((int)h_hist_data[i] > 31) {
            std::cout << h_hist_data[i];
        }
        std::cout << (int)h_hist_data[i];
        std::cout << std::endl;
    }*/
    h_bin_data_CPU = (unsigned long long*)malloc(BIN_COUNT * sizeof(unsigned long long));
    h_bin_data_GPU = (unsigned long long*)malloc(BIN_COUNT * sizeof(unsigned long long));

    hipError_t cudaStatus = histogramWithCuda(h_hist_data, h_bin_data_GPU, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "histogramWithCuda failed!");
        return 1;
    }

    myhistogramCPU(h_hist_data, h_bin_data_CPU, size);

    //std::cout << "\n0 - " << h_bin_data_CPU[0] << std::endl;
    /*for (int i = 0; i < BIN_COUNT; i++)
    {
        if (h_bin_data_CPU[i] > 0) {
            std::cout << i << " - " << h_bin_data_CPU[i] << std::endl;
        }
    }
    std::cout << std::endl;*/
    std::cout << "\nСравнение результатов...\n";
    bool match = true;
    for (size_t i = 0; i < BIN_COUNT; i++)
    {
        if (h_bin_data_CPU[i] != h_bin_data_GPU[i]) {
            printf("Index %d. Asserted: %d. From kernel: %d ", i, h_bin_data_CPU[i], h_bin_data_GPU[i]);
            printf("FAILED \n");
            match = false;
        }
        /*else {
            printf("\n");
        }*/
    }
    std::cout << (match ? "Результаты сошлись" : "Результаты не сошлись");

    //std::cout << "\nGPU:\n";
    //std::cout << 0 << " - " << h_bin_data_GPU[0] << std::endl;
    /*for (int i = 0; i < BIN_COUNT; i++)
    {
        if (h_bin_data_GPU[i] > 0) {
            std::cout << i << " - " << h_bin_data_GPU[i] << std::endl;
        }
    }*/


    return 0;

}